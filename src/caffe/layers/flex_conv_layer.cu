#include "hip/hip_runtime.h"
#include <vector>
#include <cfloat>
#include <iostream>

#include "caffe/layers/flex_conv_layer.hpp"
using namespace std;

namespace caffe {

template <typename Dtype>
__global__ void GpuGetMapping(const int nthreads, const Dtype* bottom_data,
  const int channels, const int height, const int width, const int pad_h,
  const int pad_w, const int sample_kernel_h, const int sample_kernel_w,
  Dtype* max_mapping, Dtype* min_mapping){
  CUDA_KERNEL_LOOP(index, nthreads){
    const int w = index % width;
    const int h = ( index / width ) % height;
    const int c = ( index / width / height ) % channels;
    const int n = ( index / width / height / channels );
    int hstart = h - pad_h;
    int wstart = w - pad_w;
    int hend = min(hstart + sample_kernel_h, height);
    int wend = min(wstart + sample_kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    Dtype maxval = -FLT_MAX;
    Dtype minval = FLT_MAX;
    const Dtype* bottom_slice =
      bottom_data + (n * channels + c ) * height * width;
    for (int ih = hstart; ih < hend; ++ih){
      for (int iw = wstart; iw < wend; ++iw){
        const int idx = ih * width + iw;
        // if (index == 0){
        //   printf("value = %f, index = %d\n",bottom_slice[idx], idx);
        // }

        if (bottom_slice[idx] > maxval){
          maxval = bottom_slice[idx];

        }
        if (bottom_slice[idx] < minval){
          minval = bottom_slice[idx];
        }
      }
    }
    max_mapping[index] = maxval;
    min_mapping[index] = minval;
    // if (index == 0){
    //   printf("maxvalue = %f\n", max_mapping[index]);
    //   printf("*******\n");
    // }
  }


}

template <typename Dtype>
__global__ void GpuComputeWeightMask(const int nthreads, const Dtype* weights,
    Dtype* max_bit_mask, Dtype* min_bit_mask){
      CUDA_KERNEL_LOOP(index, nthreads){
        if (weights[index] < 0.){
          min_bit_mask[index] = weights[index];
          max_bit_mask[index] = 0.;
        }else{
          min_bit_mask[index] = 0.;
          max_bit_mask[index] = weights[index];
        }
      }
}

template <typename Dtype>
__global__ void GpuComputeWeightDiff(const int nthreads, const Dtype* weight, Dtype* weight_diff,
    const Dtype* max_mask, const Dtype* min_mask){
      CUDA_KERNEL_LOOP(index, nthreads){
          if (weight[index] < 0.){
            weight_diff[index] = min_mask[index];
          }else{
            weight_diff[index] = max_mask[index];
          }
      }
}


template <typename Dtype>
void FlexConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const int wcount = this->blobs_[0]->count();



  Dtype *max_bit_mask, *min_bit_mask;
  CUDA_CHECK(hipMalloc((void **) &max_bit_mask, wcount * sizeof(Dtype)));
  CUDA_CHECK(hipMalloc((void **) &min_bit_mask, wcount * sizeof(Dtype)));


  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    const int bcount = bottom[i]->count();

    Dtype *max_mapping, *min_mapping;
    CUDA_CHECK(hipMalloc((void **) &max_mapping, bcount * sizeof(Dtype)));
    CUDA_CHECK(hipMalloc((void **) &min_mapping, bcount * sizeof(Dtype)));

    Dtype* top_data = top[i]->mutable_gpu_data();
    const int tcount = top[i]->count();
    const int images_count = top[i]->count(2);
    const int channels_count = top[i]->count(1);

    Dtype *max_top_data, *min_top_data;

    CUDA_CHECK(hipMalloc((void **) &max_top_data, tcount * sizeof(Dtype)));
    CUDA_CHECK(hipMalloc((void **) &min_top_data, tcount * sizeof(Dtype)));

    caffe_copy(tcount, top_data, max_top_data);
    caffe_copy(tcount, top_data, min_top_data);


    GpuGetMapping<Dtype><<<CAFFE_GET_BLOCKS(bcount), CAFFE_CUDA_NUM_THREADS>>>(
      bcount, bottom_data, this->channels_, height_, width_, pad_h_, pad_w_, sample_kernel_h_,
      sample_kernel_w_, max_mapping, min_mapping);


      // Dtype* maxcpu = (Dtype *)malloc(bcount * sizeof(Dtype));
      // Dtype* original = (Dtype *)malloc(bcount * sizeof(Dtype));
      // hipMemcpy(maxcpu, max_mapping, bcount * sizeof(Dtype), hipMemcpyDeviceToHost);
      // hipMemcpy(original, bottom_data, bcount * sizeof(Dtype), hipMemcpyDeviceToHost);
      // cout << "value: " << maxcpu[0] << endl;
      // for (int i = 0; i < 200; i++){
      //   if (maxcpu[0] == original[i]){
      //     cout << "index:" << i << endl;
      //   }
      // }
      // cout << maxcpu[0] << endl;
      // // cout << original[0] << " " << original[1] << " " << original[2] << " " <<
      // // original[32] << " " << original[34] << " " << original[64] << " " << original[65] << " " << original[66] << endl;
      // cout << original[0] << " " <<original[1] << " " << " " << original[2] << " " << original[3] << " " << endl;
      // cout << original[32] << " " <<original[33] << " " << " " << original[34] << " " << original[35] << " " << endl;
      // cout << "******" << endl;

    for (int n = 0; n < this->num_; ++n) {

      GpuComputeWeightMask<Dtype><<<CAFFE_GET_BLOCKS(wcount), CAFFE_CUDA_NUM_THREADS>>>(
        wcount, weight, max_bit_mask, min_bit_mask);

        // Dtype* maxcpu = (Dtype *)malloc(wcount * sizeof(Dtype));
        // Dtype* original = (Dtype *)malloc(wcount * sizeof(Dtype));
        // hipMemcpy(maxcpu, max_bit_mask, wcount * sizeof(Dtype), hipMemcpyDeviceToHost);
        // hipMemcpy(original, weight, wcount * sizeof(Dtype), hipMemcpyDeviceToHost);
        // cout << original[1] << endl;
        // cout << maxcpu[1] << endl;


      this->forward_gpu_gemm(max_mapping + n * this->bottom_dim_, max_bit_mask,
          max_top_data + n * this->top_dim_);
      this->forward_gpu_gemm(min_mapping + n * this->bottom_dim_, min_bit_mask,
          min_top_data + n * this->top_dim_);

      caffe_gpu_axpy<Dtype>(this->top_dim_, 1, max_top_data + n * this->top_dim_, min_top_data + n * this->top_dim_);
      int a = 0;
      // if (n == 0){
      //   Dtype* cpu = (Dtype *)malloc(tcount * sizeof(Dtype));
      //   hipMemcpy(cpu, min_top_data, tcount * sizeof(Dtype), hipMemcpyDeviceToHost);
      //   for (int i = 0; i < channels_count; i+=images_count){
      //     cout << "(" << cpu[i] << "," << cpu[i+1] << ")" << " ";
      //     a++;
      //   }
      //
      //   cout << "NUMS:" << a << endl;
      //
      // }

      caffe_copy(this->top_dim_, min_top_data + n * this->top_dim_, top_data + n * this->top_dim_);

      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }
    }
    hipFree(max_mapping); hipFree(min_mapping); hipFree(max_top_data); hipFree(min_top_data);
  }
  hipFree(max_bit_mask); hipFree(min_bit_mask);
}

template <typename Dtype>
void FlexConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const int wcount = reverse_dimensions() ? this->channels_ * this->blobs_[0]->count(1) :
                                            this->num_output_ * this->blobs_[0]->count(1);

  Dtype *max_mask, *min_mask;
  CUDA_CHECK(hipMalloc((void **) &max_mask, wcount * sizeof(Dtype)));
  CUDA_CHECK(hipMalloc((void **) &min_mask, wcount * sizeof(Dtype)));


  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();

  caffe_copy(wcount, weight_diff, max_mask);
  caffe_copy(wcount, weight_diff, min_mask);

  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();

    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();

      const int bcount = bottom[i]->count();
      Dtype *max_mapping, *min_mapping;
      CUDA_CHECK(hipMalloc((void **) &max_mapping, bcount * sizeof(Dtype)));
      CUDA_CHECK(hipMalloc((void **) &min_mapping, bcount * sizeof(Dtype)));

      GpuGetMapping<Dtype><<<CAFFE_GET_BLOCKS(bcount), CAFFE_CUDA_NUM_THREADS>>>(
        bcount, bottom_data, this->channels_, height_, width_, pad_h_, pad_w_, sample_kernel_h_,
        sample_kernel_w_, max_mapping, min_mapping);

      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {

          this->weight_gpu_gemm(max_mapping + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, max_mask);

          this->weight_gpu_gemm(min_mapping + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, min_mask);

          GpuComputeWeightDiff<Dtype><<<CAFFE_GET_BLOCKS(wcount), CAFFE_CUDA_NUM_THREADS>>>(
            wcount, weight, weight_diff, max_mask, min_mask);

          // this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
          //     top_diff + n * this->top_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
      hipFree(max_mapping); hipFree(min_mapping);
    }
  }
  hipFree(max_mask); hipFree(min_mask);
}

INSTANTIATE_LAYER_GPU_FUNCS(FlexConvolutionLayer);

}  // namespace caffe
